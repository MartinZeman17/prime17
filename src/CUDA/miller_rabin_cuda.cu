/**
 * @file miller_rabin_cuda.cu
 *
 * @brief This module provides functions for performing the deterministic Miller-Rabin primality test.
 *        This version of the test is deterministic all unsigned 64 bit integers.
 *        The test is accelerated on a GPU through CUDA.
 *        Run the test on a wide rangle of numbers for maximizing perfoormance.
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <limits.h>
#include <assert.h>
#include <chrono>


/**
 * @brief Performs the square-and-multiply algorithm to calculate a**b % m.
 *
 * This function is executed on the GPU device.
 * This function works properly for all unsigned 64-bit integers - it uses 128-bit integers to prevent overflow.
 *
 * @param a The base value.
 * @param b The exponent value.
 * @param m The modulus value.
 *
 * @return The result of a^b mod m.
 */
__device__ u_int64_t squareAndMultiply(u_int64_t a, u_int64_t b, u_int64_t m) {
    a = a % m;  // Reduce a modulo m to ensure a is within the range [0, m-1]
    u_int64_t result = 1;

    while (b > 0) {
        if (b & 1) {
            // Multiply result with a, and then reduce modulo m
            result = ((__uint128_t)result * (__uint128_t)a) % m;
        }
        // Square a, and then reduce modulo m
        a = ((__uint128_t)a * (__uint128_t)a) % m;
        // Right shift b by 1 (equivalent to dividing by 2)
        b >>= 1;
        
    }

    return result;
}




/**
 * @brief Performs the Miller-Rabin primality test to check if a number is probably prime.
 *
 * This test is deterministic for all unsigned 64-bit integers.
 * This function is executed on the GPU device.
 *
 * @param n The number to be tested for primality.
 *
 * @return true if n is probably prime, false if n is composite.
 */
__device__ bool millerRabin(u_int64_t n) {

    // Predefined bases for uint64_t
    u_int8_t BASE[] = {2, 3, 5, 7, 11, 13, 17, 19, 23, 29, 31, 37, 41};

    // Check if n is a small prime number
    if (n <= 41) {
        if (n <= 1) {
            return false;
        }
        for (int i = 0; i < 13; i++) {
            if (n == BASE[i]) {
                return true;
            }
        }
    }

    // Check if n is divisible by 2
    if (n % 2 == 0) {
        return false;
    }

    // Compute n - 1 = 2^s * d
    u_int64_t d = n - 1;
    u_int64_t s = 0;
    while ((d & 1) == 0) {
        d >>= 1;
        s++;
    }


    // Perform the Miller-Rabin test with the predefined bases
    for (int i = 0; i < 13; i++) {
        u_int64_t a = BASE[i];
        u_int64_t x = squareAndMultiply(a, d, n);

        if (x == 1 || x == n - 1) {
            // Continue to the next base if x is a probable witness
            continue;
        }

        bool continueLoop = false;

        for (int j = 0; j < s - 1; j++) {
            x = squareAndMultiply(x, 2, n);

            if (x == 1) {
                // n is composite, return false
                return false;
            }

            if (x == n - 1) {
                // x is a probable witness, continue to the next base
                continueLoop = true;
                break;
            }
        }

        if (continueLoop) {
            continue;
        }

        // x is not a probable witness, n is composite
        return false;
    }

    // n passed the Miller-Rabin test for all bases, it is probably prime
    return true;
}


/**
 * @brief CUDA kernel function for parallel prime testing using the Miller-Rabin algorithm.
 *
 * This function is executed on the GPU device.
 *
 * @param boolArray A pointer to the boolean array indicating whether each number is prime.
 * @param primes A pointer to the array of numbers to be tested for primality.
 * @param size The size of the array.
 */
__global__ void kernel(bool* boolArray, const u_int64_t* primes, const u_int64_t size) {
    // Calculate the global thread ID
    u_int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread ID is within the valid range
    if (tid < size) {
        // Test the primality of the number at the current thread ID
        boolArray[tid] = millerRabin(primes[tid]);
    }
}



/**
 * @brief Tests whether each number in the given array of numbers is prime.
 *
 * This function utilizes CUDA to parallelize the prime testing on the GPU.
 *
 * @param numbers A pointer to an array of numbers to be tested for primality.
 * @param size The size of the array.
 *
 * @return A pointer to a bool array indicating whether each number is prime.
 *         The caller is responsible for freeing the memory allocated for the bool array.
 */
bool* testPrimes(u_int64_t* numbers, size_t size) {
    u_int64_t* d_numbers;   // Device memory pointer for numbers
    bool* d_boolArray;      // Device memory pointer for bool array

    // Allocate device memory for numbers and bools
    hipMalloc((void**)&d_numbers, sizeof(u_int64_t) * size);
    hipMalloc((void**)&d_boolArray, sizeof(bool) * size);

    // Copy the numbers from host to device memory
    hipMemcpy(d_numbers, numbers, sizeof(u_int64_t) * size, hipMemcpyHostToDevice);

    // Allocate host memory for the resulting bool array
    bool* boolArray = (bool*)malloc(sizeof(bool) * size);

    // Set the block size for CUDA kernel execution and calculate the grid size
    constexpr u_int64_t BLOCKSIZE = 256;
    u_int64_t gridSize = (size + BLOCKSIZE - 1) / BLOCKSIZE;

    // Launch the CUDA kernel to test the primality of numbers
    kernel<<<gridSize, BLOCKSIZE>>>(d_boolArray, d_numbers, size);

    // Copy the bool array from device to host memory
    hipMemcpy(boolArray, d_boolArray, sizeof(bool) * size, hipMemcpyDeviceToHost);

    // Free the allocated device memory
    hipFree(d_numbers);
    hipFree(d_boolArray);

    return boolArray;
}

int test_from_file(const char* filename, bool correctResultValue)
{
    // load the test data
    std::cout << "loading testing data..." << std::endl;
    FILE *file = fopen(filename, "r");
    if (file == NULL) {
        perror("Error opening file");
        return 1;
    }

    u_int64_t *primes = NULL;
    u_int64_t number;
    size_t size = 0;
    while (fscanf(file, "%llu\n", &number) == 1) {
        primes = (u_int64_t *)realloc(primes, (size + 1) * sizeof(u_int64_t));
        primes[size++] = number;
    }
    fclose(file);
    std::cout << "about to test " << size << " numbers" << std::endl;

    // time function and print results
    auto start = std::chrono::high_resolution_clock::now();
    bool* result = testPrimes(primes, size);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = end - start;

    // results
    printf("testing took: %.3f sesonds\n\n", duration / 1000);

    // test correctness
    for (int i = 0; i < size; i++) {
        assert (result[i] == correctResultValue);
    }

    // free memory
    free(result);
    free(primes);

    return 0;
}



int main() {

    // load the test data
    std::cout << "note - first warmup run is expected to be longer" << std::endl;
    std::cout << "testing over non primes" << std::endl;
    test_from_file("./data/non_primes.csv", false);

    std::cout << "testing over pseudo primes" << std::endl;
    test_from_file("./data/pseudo_primes.csv", false);

    std::cout << "testing over edge cases" << std::endl;
    test_from_file("./data/edge_cases_false.csv", false);
    test_from_file("./data/edge_cases_true.csv", true);

    std::cout << "testing over sieve of erathostenes for all unsigned numbers < 2**20" << std::endl;
    test_from_file("./data/primes_2**20.csv", true);

    std::cout << "testing over primes ~ 2**43" << std::endl;
    test_from_file("./data/primes_2**43.csv", true);

    std::cout << "testing over primes ~ 2**63" << std::endl;
    test_from_file("./data/primes_2**63.csv", true);

    return 0;
}
